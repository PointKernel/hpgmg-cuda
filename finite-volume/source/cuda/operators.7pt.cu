#include "hip/hip_runtime.h"
/*
# Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
#
# Redistribution and use in source and binary forms, with or without
# modification, are permitted provided that the following conditions
# are met:
#  * Redistributions of source code must retain the above copyright
#    notice, this list of conditions and the following disclaimer.
#  * Redistributions in binary form must reproduce the above copyright
#    notice, this list of conditions and the following disclaimer in the
#    documentation and/or other materials provided with the distribution.
#  * Neither the name of NVIDIA CORPORATION nor the names of its
#    contributors may be used to endorse or promote products derived
#    from this software without specific prior written permission.
#
# THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
# EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
# IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
# PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
# CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
# EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
# PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
# PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
# OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
# (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
# OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
//------------------------------------------------------------------------------------------------------------------------------
// Nikolay Sakharnykh
// nsakharnykh@nvidia.com
// Copyright (c) 2014-2015, NVIDIA CORPORATION.  All rights reserved.
//------------------------------------------------------------------------------------------------------------------------------
// Samuel Williams
// SWWilliams@lbl.gov
// Lawrence Berkeley National Lab
//------------------------------------------------------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
//------------------------------------------------------------------------------------------------------------------------------
#ifdef _OPENMP
#include <omp.h>
#endif
//------------------------------------------------------------------------------------------------------------------------------
#include "../timers.h"
#include "../defines.h"
#include "../level.h"
#include "../operators.h"
//------------------------------------------------------------------------------------------------------------------------------
#define STENCIL_VARIABLE_COEFFICIENT
//------------------------------------------------------------------------------------------------------------------------------
// below are stencil operators versions using different memory load options
//------------------------------------------------------------------------------------------------------------------------------
#ifndef CUDA_STENCIL_OPT_TEX
//------------------------------------------------------------------------------------------------------------------------------
// calculate Dinv?
#ifdef STENCIL_VARIABLE_COEFFICIENT
  #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz ...
  #define calculate_Dinv()                                      \
  (                                                             \
    1.0 / (a*alpha[ijk] - b*h2inv*(                             \
             + beta_i[ijk        ]*( valid[ijk-1      ] - 2.0 ) \
             + beta_j[ijk        ]*( valid[ijk-jStride] - 2.0 ) \
             + beta_k[ijk        ]*( valid[ijk-kStride] - 2.0 ) \
             + beta_i[ijk+1      ]*( valid[ijk+1      ] - 2.0 ) \
             + beta_j[ijk+jStride]*( valid[ijk+jStride] - 2.0 ) \
             + beta_k[ijk+kStride]*( valid[ijk+kStride] - 2.0 ) \
          ))                                                    \
  )
  #else // variable coefficient Poisson ...
  #define calculate_Dinv()                                      \
  (                                                             \
    1.0 / ( -b*h2inv*(                                          \
             + beta_i[ijk        ]*( valid[ijk-1      ] - 2.0 ) \
             + beta_j[ijk        ]*( valid[ijk-jStride] - 2.0 ) \
             + beta_k[ijk        ]*( valid[ijk-kStride] - 2.0 ) \
             + beta_i[ijk+1      ]*( valid[ijk+1      ] - 2.0 ) \
             + beta_j[ijk+jStride]*( valid[ijk+jStride] - 2.0 ) \
             + beta_k[ijk+kStride]*( valid[ijk+kStride] - 2.0 ) \
          ))                                                    \
  )
  #endif
#else // constant coefficient case... 
  #define calculate_Dinv()          \
  (                                 \
    1.0 / (a - b*h2inv*(            \
             + valid[ijk-1      ]   \
             + valid[ijk-jStride]   \
             + valid[ijk-kStride]   \
             + valid[ijk+1      ]   \
             + valid[ijk+jStride]   \
             + valid[ijk+kStride]   \
             - 12.0                 \
          ))                        \
  )
#endif

#if defined(STENCIL_FUSE_DINV) && defined(STENCIL_FUSE_BC)
#define Dinv_ijk() calculate_Dinv() // recalculate it
#else
#define Dinv_ijk() Dinv[ijk]        // simply retrieve it rather than recalculating it
#endif
//------------------------------------------------------------------------------------------------------------------------------
#ifdef STENCIL_FUSE_BC

  #ifdef STENCIL_VARIABLE_COEFFICIENT
    #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz ...
    #define apply_op_ijk(x)                                                                   \
    (                                                                                         \
      a*alpha[ijk]*x[ijk]                                                                     \
      -b*h2inv*(                                                                              \
        + beta_i[ijk        ]*( valid[ijk-1      ]*( x[ijk] + x[ijk-1      ] ) - 2.0*x[ijk] ) \
        + beta_j[ijk        ]*( valid[ijk-jStride]*( x[ijk] + x[ijk-jStride] ) - 2.0*x[ijk] ) \
        + beta_k[ijk        ]*( valid[ijk-kStride]*( x[ijk] + x[ijk-kStride] ) - 2.0*x[ijk] ) \
        + beta_i[ijk+1      ]*( valid[ijk+1      ]*( x[ijk] + x[ijk+1      ] ) - 2.0*x[ijk] ) \
        + beta_j[ijk+jStride]*( valid[ijk+jStride]*( x[ijk] + x[ijk+jStride] ) - 2.0*x[ijk] ) \
        + beta_k[ijk+kStride]*( valid[ijk+kStride]*( x[ijk] + x[ijk+kStride] ) - 2.0*x[ijk] ) \
      )                                                                                       \
    )
    #else // variable coefficient Poisson ...
    #define apply_op_ijk(x)                                                                   \
    (                                                                                         \
      -b*h2inv*(                                                                              \
        + beta_i[ijk        ]*( valid[ijk-1      ]*( x[ijk] + x[ijk-1      ] ) - 2.0*x[ijk] ) \
        + beta_j[ijk        ]*( valid[ijk-jStride]*( x[ijk] + x[ijk-jStride] ) - 2.0*x[ijk] ) \
        + beta_k[ijk        ]*( valid[ijk-kStride]*( x[ijk] + x[ijk-kStride] ) - 2.0*x[ijk] ) \
        + beta_i[ijk+1      ]*( valid[ijk+1      ]*( x[ijk] + x[ijk+1      ] ) - 2.0*x[ijk] ) \
        + beta_j[ijk+jStride]*( valid[ijk+jStride]*( x[ijk] + x[ijk+jStride] ) - 2.0*x[ijk] ) \
        + beta_k[ijk+kStride]*( valid[ijk+kStride]*( x[ijk] + x[ijk+kStride] ) - 2.0*x[ijk] ) \
      )                                                                                       \
    )
    #endif
  #else  // constant coefficient case...  
    #define apply_op_ijk(x)                                \
    (                                                    \
      a*x[ijk] - b*h2inv*(                               \
        + valid[ijk-1      ]*( x[ijk] + x[ijk-1      ] ) \
        + valid[ijk-jStride]*( x[ijk] + x[ijk-jStride] ) \
        + valid[ijk-kStride]*( x[ijk] + x[ijk-kStride] ) \
        + valid[ijk+1      ]*( x[ijk] + x[ijk+1      ] ) \
        + valid[ijk+jStride]*( x[ijk] + x[ijk+jStride] ) \
        + valid[ijk+kStride]*( x[ijk] + x[ijk+kStride] ) \
                       -12.0*( x[ijk]                  ) \
      )                                                  \
    )
  #endif // variable/constant coefficient

#endif

//------------------------------------------------------------------------------------------------------------------------------
#ifndef STENCIL_FUSE_BC

  #ifdef STENCIL_VARIABLE_COEFFICIENT
    #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz...
    #define apply_op_ijk(x)                               \
    (                                                     \
      a*alpha[ijk]*x[ijk]                                 \
     -b*h2inv*(                                           \
        + beta_i[ijk+1      ]*( x[ijk+1      ] - x[ijk] ) \
        + beta_i[ijk        ]*( x[ijk-1      ] - x[ijk] ) \
        + beta_j[ijk+jStride]*( x[ijk+jStride] - x[ijk] ) \
        + beta_j[ijk        ]*( x[ijk-jStride] - x[ijk] ) \
        + beta_k[ijk+kStride]*( x[ijk+kStride] - x[ijk] ) \
        + beta_k[ijk        ]*( x[ijk-kStride] - x[ijk] ) \
      )                                                   \
    )
    #else // variable coefficient Poisson...
    #define apply_op_ijk(x)                               \
    (                                                     \
      -b*h2inv*(                                          \
        + beta_i[ijk+1      ]*( x[ijk+1      ] - x[ijk] ) \
        + beta_i[ijk        ]*( x[ijk-1      ] - x[ijk] ) \
        + beta_j[ijk+jStride]*( x[ijk+jStride] - x[ijk] ) \
        + beta_j[ijk        ]*( x[ijk-jStride] - x[ijk] ) \
        + beta_k[ijk+kStride]*( x[ijk+kStride] - x[ijk] ) \
        + beta_k[ijk        ]*( x[ijk-kStride] - x[ijk] ) \
      )                                                   \
    )
    #endif
  #else  // constant coefficient case...  
    #define apply_op_ijk(x)            \
    (                                \
      a*x[ijk] - b*h2inv*(           \
        + x[ijk+1      ]             \
        + x[ijk-1      ]             \
        + x[ijk+jStride]             \
        + x[ijk-jStride]             \
        + x[ijk+kStride]             \
        + x[ijk-kStride]             \
        - x[ijk        ]*6.0         \
      )                              \
    )
  #endif // variable/constant coefficient

#endif // BCs
//------------------------------------------------------------------------------------------------------------------------------
#else  // stencil-opt-tex
//------------------------------------------------------------------------------------------------------------------------------
// calculate Dinv?
#ifdef STENCIL_VARIABLE_COEFFICIENT
  #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz ...
  #define calculate_Dinv()                                      \
  (                                                             \
    1.0 / (a*alpha[ijk] - b*h2inv*(                             \
             + beta_i[ijk        ]*( __ldg(valid + ijk-1      ) - 2.0 ) \
             + beta_j[ijk        ]*( __ldg(valid + ijk-jStride) - 2.0 ) \
             + beta_k_cur         *( valid_kprev                - 2.0 ) \
             + beta_i[ijk+1      ]*( __ldg(valid + ijk+1      ) - 2.0 ) \
             + beta_j[ijk+jStride]*( __ldg(valid + ijk+jStride) - 2.0 ) \
             + beta_k_next        *( valid_knext                - 2.0 ) \
          ))                                                    \
  )
  #else // variable coefficient Poisson ...
  // optimization: 
  //   use registers to cache valid along k dim for valid and beta_k
  //   use ldg to cache along i-j plane for valid
  #define calculate_Dinv()                                      \
  (                                                             \
    1.0 / ( -b*h2inv*(                                          \
             + beta_i[ijk        ]*( __ldg(valid + ijk-1      ) - 2.0 ) \
             + beta_j[ijk        ]*( __ldg(valid + ijk-jStride) - 2.0 ) \
             + beta_k_cur         *( valid_kprev                - 2.0 ) \
             + beta_i[ijk+1      ]*( __ldg(valid + ijk+1      ) - 2.0 ) \
             + beta_j[ijk+jStride]*( __ldg(valid + ijk+jStride) - 2.0 ) \
             + beta_k_next        *( valid_knext                - 2.0 ) \
          ))                                                    \
  )
  #endif
#else // constant coefficient case... 
  #define calculate_Dinv()          \
  (                                 \
    1.0 / (a - b*h2inv*(            \
             + valid[ijk-1      ]   \
             + valid[ijk-jStride]   \
             + valid[ijk-kStride]   \
             + valid[ijk+1      ]   \
             + valid[ijk+jStride]   \
             + valid[ijk+kStride]   \
             - 12.0                 \
          ))                        \
  )
#endif

#if defined(STENCIL_FUSE_DINV) && defined(STENCIL_FUSE_BC)
#define Dinv_ijk() calculate_Dinv() // recalculate it
#else
#define Dinv_ijk() Dinv[ijk]        // simply retriev it rather than recalculating it
#endif
//------------------------------------------------------------------------------------------------------------------------------
#ifdef STENCIL_FUSE_BC

  #ifdef STENCIL_VARIABLE_COEFFICIENT
    #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz ...
    #define apply_op_ijk(x)                                                                   \
    (                                                                                         \
      a*alpha[ijk]*x_cur                                                                      \
      -b*h2inv*(                                                                              \
        + beta_i[ijk        ]*( __ldg(valid + ijk-1      )*( x_cur  + __ldg(x + ijk-1      ) ) - 2.0*x_cur ) \
        + beta_j[ijk        ]*( __ldg(valid + ijk-jStride)*( x_cur  + __ldg(x + ijk-jStride) ) - 2.0*x_cur ) \
        + beta_k_cur         *( valid_kprev               *( x_cur  + x_kprev                ) - 2.0*x_cur ) \
        + beta_i[ijk+1      ]*( __ldg(valid + ijk+1      )*( x_cur  + __ldg(x + ijk+1      ) ) - 2.0*x_cur ) \
        + beta_j[ijk+jStride]*( __ldg(valid + ijk+jStride)*( x_cur  + __ldg(x + ijk+jStride) ) - 2.0*x_cur ) \
        + beta_k_next        *( valid_knext               *( x_cur  + x_knext                ) - 2.0*x_cur ) \
      )                                                                                       \
    )
    #else // variable coefficient Poisson ...
    // optimizations: 
    //   use registers to cache points along k dim for x, valid and beta_k
    //   use ldg to cache along i-j plane for x and valid
    #define apply_op_ijk(x)                                                                   \
    (                                                                                         \
      -b*h2inv*(                                                                              \
        + beta_i[ijk        ]*( __ldg(valid + ijk-1      )*( x_cur  + __ldg(x + ijk-1      ) ) - 2.0*x_cur ) \
        + beta_j[ijk        ]*( __ldg(valid + ijk-jStride)*( x_cur  + __ldg(x + ijk-jStride) ) - 2.0*x_cur ) \
        + beta_k_cur         *( valid_kprev               *( x_cur  + x_kprev                ) - 2.0*x_cur ) \
        + beta_i[ijk+1      ]*( __ldg(valid + ijk+1      )*( x_cur  + __ldg(x + ijk+1      ) ) - 2.0*x_cur ) \
        + beta_j[ijk+jStride]*( __ldg(valid + ijk+jStride)*( x_cur  + __ldg(x + ijk+jStride) ) - 2.0*x_cur ) \
        + beta_k_next        *( valid_knext               *( x_cur  + x_knext                ) - 2.0*x_cur ) \
      )                                                                                       \
    )
    #endif
  #else  // constant coefficient case...  
    #define apply_op_ijk(x)                                \
    (                                                    \
      a*x[ijk] - b*h2inv*(                               \
        + valid[ijk-1      ]*( x[ijk] + x[ijk-1      ] ) \
        + valid[ijk-jStride]*( x[ijk] + x[ijk-jStride] ) \
        + valid[ijk-kStride]*( x[ijk] + x[ijk-kStride] ) \
        + valid[ijk+1      ]*( x[ijk] + x[ijk+1      ] ) \
        + valid[ijk+jStride]*( x[ijk] + x[ijk+jStride] ) \
        + valid[ijk+kStride]*( x[ijk] + x[ijk+kStride] ) \
                       -12.0*( x[ijk]                  ) \
      )                                                  \
    )
  #endif // variable/constant coefficient

#endif

//------------------------------------------------------------------------------------------------------------------------------
#ifndef STENCIL_FUSE_BC

  #ifdef STENCIL_VARIABLE_COEFFICIENT
    #ifdef USE_HELMHOLTZ // variable coefficient Helmholtz...
    #define apply_op_ijk(x)                               \
    (                                                     \
      a*alpha[ijk]*x_cur                                  \
     -b*h2inv*(                                           \
        + beta_i[ijk+1      ]*( __ldg(x + ijk+1      ) - x_cur ) \
        + beta_i[ijk        ]*( __ldg(x + ijk-1      ) - x_cur ) \
        + beta_j[ijk+jStride]*( __ldg(x + ijk+jStride) - x_cur ) \
        + beta_j[ijk        ]*( __ldg(x + ijk-jStride) - x_cur ) \
        + beta_k_next        *(                x_knext - x_cur ) \
        + beta_k_cur         *(                x_kprev - x_cur ) \
      )                                                   \
    )
    #else // variable coefficient Poisson...
    // optimizations: 
    //   use registers to cache points along k dim for x and beta_k
    //   use ldg to cache along i-j plane for x
    #define apply_op_ijk(x)                               \
    (                                                     \
      -b*h2inv*(                                          \
        + beta_i[ijk+1      ]*( __ldg(x + ijk+1      ) - x_cur ) \
        + beta_i[ijk        ]*( __ldg(x + ijk-1      ) - x_cur ) \
        + beta_j[ijk+jStride]*( __ldg(x + ijk+jStride) - x_cur ) \
        + beta_j[ijk        ]*( __ldg(x + ijk-jStride) - x_cur ) \
        + beta_k_next        *(                x_knext - x_cur ) \
        + beta_k_cur         *(                x_kprev - x_cur ) \
      )                                                   \
    )
    #endif
  #else  // constant coefficient case...  
    #define apply_op_ijk(x)            \
    (                                \
      a*x[ijk] - b*h2inv*(           \
        + x[ijk+1      ]             \
        + x[ijk-1      ]             \
        + x[ijk+jStride]             \
        + x[ijk-jStride]             \
        + x[ijk+kStride]             \
        + x[ijk-kStride]             \
        - x[ijk        ]*6.0         \
      )                              \
    )
  #endif // variable/constant coefficient

#endif // BCs
#endif // stencil-opt-tex

//------------------------------------------------------------------------------------------------------------------------------
#ifdef  USE_GSRB
#define NUM_SMOOTHS      2 // RBRB
#include "gsrb.h"
#elif   USE_CHEBY
#define NUM_SMOOTHS      1
#define CHEBYSHEV_DEGREE 4 // i.e. one degree-4 polynomial smoother
#include "chebyshev.h"
#else
#error You must compile CUDA code with either -DUSE_GSRB or -DUSE_CHEBY, other smoothers are not currently supported
#endif
//------------------------------------------------------------------------------------------------------------------------------
#include "residual.h"
#include "blockCopy.h"
#include "misc.h"
#include "boundary_fd.h"
#include "restriction.h"
#include "interpolation.h"
//------------------------------------------------------------------------------------------------------------------------------
